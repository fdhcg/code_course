#include "hip/hip_runtime.h"
#include<iostream>
#include<time.h>
#include "hip/hip_runtime.h"
#include ""
#include"hip/device_functions.h"
using namespace std;
double *generator(int Num);
#define N 8000;
__global__ void convByGPU(double *A, double *out, double *window) {
	__shared__ double temp[16 * 16];
	int Num = N;
	int i;
	const int xIndex = blockDim.x*blockIdx.x + threadIdx.x;
	const int yIndex = blockDim.y*blockIdx.y + threadIdx.y;
	if (xIndex < Num&&yIndex < Num) {
		double convW[9] = { A[Num*xIndex + yIndex],A[Num*xIndex + yIndex + 1],A[Num*xIndex + yIndex + 2],A[Num*(xIndex + 1) + yIndex],A[Num*(xIndex + 1) + yIndex + 1],A[Num*(xIndex + 1) + yIndex + 2],
						A[Num*(xIndex + 2) + yIndex],A[Num*(xIndex + 2) + yIndex + 1],A[Num*(xIndex + 2) + yIndex + 2] };
		temp[threadIdx.y + threadIdx.x * 16] = 0.0;
		for (i = 0; i < 9; i++) {
			temp[threadIdx.x * 16 + threadIdx.y] += convW[i] * window[i];
		}
		__syncthreads();
		i = 2;
		const int temp_index = threadIdx.x * 16 + threadIdx.y;
		while (i <= 16 * 16) {
			if (temp_index%i == 0) {
				if (temp[temp_index] < temp[temp_index + i / 2])
					temp[temp_index] = temp[temp_index + i / 2];
			}
			__syncthreads();
			i = i * 2;
		}
		out[blockIdx.x*Num / 16 + blockIdx.y] = temp[0];
	}


}
class convfunc {
private:
	double window1[9] = { 1.0 / 9.0,1.0 / 9.0,1.0 / 9.0,1.0 / 9.0,1.0 / 9.0,1.0 / 9.0,1.0 / 9.0,1.0 / 9.0,1.0 / 9.0 };
	double window2[9] = { 1.0 / 12.0,1.0 / 6.0,1.0 / 12.0,1.0 / 6.0,1.0 / 3.0,1.0 / 6.0,1 / 12.0,1.0 / 6.0,1.0 / 12.0 };
	double window3[9] = { 0,1,0,-1,0,-1,-1,1,0 };
public:
	void convCPU(double *A, double *out) {
		int i, j, m, n;
		double tmp;
		int Num = N;
		double *temp = new double[Num*Num];
		clock_t time[2] = {};
		time[0] = clock();
		for (i = 0; i < Num; i++) {
			for (j = 0; j < Num; j++) {
				temp[Num*i + j] = 0;
				double convW[9] = { A[Num*i + j],A[Num*i + j + 1],A[Num*i + j + 2],A[Num*(i + 1) + j],A[Num*(i + 1) + j + 1],A[Num*(i + 1) + j + 2],
					A[Num*(i + 2) + j],A[Num*(i + 2) + j + 1],A[Num*(i + 2) + j + 2] };
				for (m = 0; m < 9; m++) {
					temp[Num*i + j] += convW[m] * window2[m];
				}

			}

		}
		for (i = 0; i < Num / 16; i++) {
			for (j = 0; j < Num / 16; j++) {
				tmp = -99;
				for (m = 0; m < 16; m++) {
					for (n = 0; n < 16; n++) {
						if (temp[16 * i + m + (16 * j + n)*Num] > tmp)
							tmp = temp[16 * i + m + (16 * j + n)*Num];

					}
				}
				out[i + j * Num / 16] = tmp;

			}
		}
		time[1] = clock();
		cout << "cpu time :" << double(time[1] - time[0]) / CLOCKS_PER_SEC << endl;
	}
	void convGPU(double *A, double *out) {
		int Num = N;
		double *cuda_A;
		clock_t time[2] = {};
		double *cuda_out, *cuda_window;
		time[0] = clock();
		dim3 dimGrid(512, 512);
		dim3 dimBlock(16, 16);
		hipMalloc((void**)&cuda_A, sizeof(double)*(Num + 2)*(Num + 2));
		hipMalloc((void**)&cuda_out, sizeof(double)*Num / 16 * Num / 16);
		hipMalloc((void**)&cuda_window, sizeof(double) * 9);
		hipMemcpy(cuda_A, A, sizeof(double)*(Num + 2)*(Num + 2), hipMemcpyHostToDevice);
		hipMemcpy(cuda_window, window2, sizeof(double) * 9, hipMemcpyHostToDevice);
		convByGPU << <dimGrid, dimBlock >> > (cuda_A, cuda_out, cuda_window);
		hipMemcpy(out, cuda_out, sizeof(double)*Num / 16 * Num / 16, hipMemcpyDeviceToHost);
		time[1] = clock();
		hipFree(cuda_A);
		hipFree(cuda_out);
		hipFree(cuda_window);
		cout << "gpu time :" << double(time[1] - time[0]) / CLOCKS_PER_SEC << endl;

	}

};

double *generator(int Num) {
	int m, n;
	Num += 2;
	double *matrixA = new double[Num*Num];
	for (m = 0; m < Num; m++) {
		for (n = 0; n < Num; n++) {
			matrixA[Num * m + n] = double(n + m) / Num * 100 + (double)(rand() % 94251473) / 194267847;
		}

	}
	return matrixA;

}
int main() {
	hipDeviceProp_t deviceProp;
	int deviceCount;
	hipError_t hipError_t;
	hipError_t = hipGetDeviceCount(&deviceCount);
	for (int i = 0; i < deviceCount; i++) {
		hipError_t = hipGetDeviceProperties(&deviceProp, i);
		cout << "设备 " << i + 1 << " 的主要属性： " << endl;
		cout << "设备显卡型号： " << deviceProp.name << endl;
		cout << "设备全局内存总量（以MB为单位）： " << deviceProp.totalGlobalMem / 1024 / 1024 << endl;
		cout << "线程块（Block）中可用的最大共享内存（以KB为单位）： " << deviceProp.sharedMemPerBlock / 1024 << endl;
		cout << "线程块（Block）种可用的32位寄存器数量： " << deviceProp.regsPerBlock << endl;
		cout << "线程块（Block）可包含的最大线程数量： " << deviceProp.maxThreadsPerBlock << endl;
		cout << "设备的计算功能集（Compute Capability）的版本号： " << deviceProp.major << "." << deviceProp.minor << endl;
		cout << "设备上多处理器的数量： " << 192 * deviceProp.multiProcessorCount << endl;
		cout << "设备线程束大小： " << deviceProp.warpSize << endl;
		cout << "GPU时钟频率(以KHz为单位)： " << deviceProp.clockRate << endl;
	}


	int i, j;
	int Num = N;
	double *matrixA = generator(Num);
	double err_mean, err_sum = 0.0;
	convfunc op;
	double *outMatrixCPU = new double[Num / 16 * Num / 16];
	double *outMatrixGPU = new double[Num / 16 * Num / 16];
	op.convCPU(matrixA, outMatrixCPU);
	op.convGPU(matrixA, outMatrixGPU);

	for (i = 0, err_sum = 0; i < Num / 16; i++) {
		for (j = 0; j < Num / 16; j++) {
			if (outMatrixCPU[i*Num / 16 + j] != 0) {
				err_sum += fabs(outMatrixCPU[i*Num / 16 + j] - outMatrixGPU[i*Num / 16 + j]) / fabs(outMatrixCPU[i*Num / 16 + j]);
			}
			else { err_sum += fabs(outMatrixCPU[i*Num / 16 + j] - outMatrixGPU[i*Num / 16 + j]); }
		}
	}
	err_mean = err_sum / (Num*Num / 16 / 16);
	printf("计算平均误差:%g\n", err_mean);

}